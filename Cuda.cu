#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

#include "Cuda.hpp"
#include "Vec3.hpp"
#include "Renderer.hpp"

using namespace std;

__global__ void raytraceCudaKernel() {
    Vec3 v(1, 2, 3);
    
    RayTracer<CPURayTracer> raytracer;
}


void launchCudaKernel() {
    dim3 gridSize(1, 1, 1);
    dim3 blockSize(1, 1, 1);
    
    raytraceCudaKernel<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
}


