#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>

#include "Cuda.hpp"
#include "Vec3.hpp"
#include "Renderer.hpp"

using namespace std;

#define BLOCK_WIDTH 4
#define BLOCK_HEIGHT 4
#define THREADS_IN_BLOCK BLOCK_WIDTH * BLOCK_HEIGHT

struct GPURayTracer {
    Scene* scene;
    
    CUDA_CALLABLE GPURayTracer(Scene* scene_) : scene(scene_) { }
    
    CUDA_CALLABLE Intersection<Triangle> findClosestIntersectedTriangle(const Ray& ray, const Shape* lastReflection) {
        Intersection<Triangle> closestIntersection;
        Intersection<Triangle> triIntersection;
        
        for(Triangle* tri = scene->triangles.begin(); tri != scene->triangles.end(); ++tri) {
            if(tri != lastReflection && tri->calculateRayIntersections(ray, &triIntersection) > 0) {
                //if(triIntersection.distanceFromRayStartSquared < closestIntersection.distanceFromRayStartSquared)
                //    closestIntersection = triIntersection;
                
                
                closestIntersection = minimum(triIntersection, closestIntersection);
            }
        }
        
        return closestIntersection;
    }
    
    CUDA_CALLABLE Intersection<Sphere> findClosestIntersectedSphere(const Ray& ray, const Shape* lastReflection) {
        Intersection<Sphere> closestIntersection;
        Intersection<Sphere> sphereIntersections[2];
        
        //printf("Total triangles: %d\n", (int)scene->triangles.total);
        
        for(Sphere* sphere = scene->spheres.begin(); sphere != scene->spheres.end(); ++sphere) {
            if(sphere == lastReflection)
                continue;
            
            int count = sphere->calculateRayIntersections(ray, sphereIntersections);
            
            if(count == 1) {
                closestIntersection = minimum(closestIntersection, sphereIntersections[0]);
            }
            else if(count == 2) {
                closestIntersection = minimum(closestIntersection, minimum(sphereIntersections[0], sphereIntersections[1]));
            }
        }
        
        return closestIntersection;
    }
    
    CUDA_CALLABLE Intersection<Shape> findClosestIntersectedShape(const Ray& ray, const Shape* lastReflection) {
        return minimum(
            findClosestIntersectedTriangle(ray, lastReflection).toGenericShapeIntersection(),
            findClosestIntersectedSphere(ray, lastReflection).toGenericShapeIntersection()
        );
    }
};

typedef RayTracer<GPURayTracer> Tracer;

std::vector<void*> cudaAllocatedMemory;

void cudaMemoryCleanup() {
    for(int i = 0; i < cudaAllocatedMemory.size(); ++i) {
        printf("Freed %llx\n", (long long)cudaAllocatedMemory[i]);
        hipFree(cudaAllocatedMemory[i]);
    }
    
    hipDeviceReset();
}

template<typename T>
void attemptCudaMalloc(T dest, size_t size) {
    if(hipMalloc(dest, size) == hipErrorOutOfMemory) {
        cudaMemoryCleanup();
        throw "Failed to allocate memory on device";
    }
    
    cudaAllocatedMemory.push_back((void *)*dest);
}

void Renderer::initializeCuda(float angle, int w, int h) {
    screenW = w;
    screenH = h;
    
    attemptCudaMalloc(&frameBuffer, sizeof(Color) * w * h);
    
    viewAngle = angle;
    distToScreen = (w / 2 ) / tan(degToRadians(angle / 2));
}

__global__ void raytraceCudaKernel(Tracer* tracer) {
    //printf("Tracer dim: %d %d\n", tracer->renderer.screenW, tracer->renderer.screenH);
    
    //printf("Total triangles: %d\n", tracer->scene.triangles.total);
    
    //tracer->raytrace();
    tracer->raytraceSingleRay(blockIdx.x, blockIdx.y);
}

template<typename T>
GeometryList<T> copyGeometryListToGPU(GeometryList<T> hostList) {
    GeometryList<T> deviceList;
    
    size_t size = sizeof(T) * hostList.total;
    
    printf("Transfered size: %d (%d)\n", (int)size, (int)sizeof(T));
    
    attemptCudaMalloc(&deviceList.list, size);
    hipMemcpy(deviceList.list, hostList.list, size, hipMemcpyHostToDevice);
    
    deviceList.total = hostList.total;
    
    return deviceList;
}

CUDATriangleList convertTriangleGeometryListToCUDATriangleList(GeometryList<Triangle>& triangles) {
    CUDATriangleList deviceList;
    
    GeometryList<float> triangleGeometry(triangles.total * cudaTriangleListSize());
    GeometryList<Material> materials(triangles.total);
    GeometryList<Color> colors(triangles.total);
    
    for(int i = 0; i < triangles.total; ++i) {
        Triangle& tri = triangles.list[i];
        float* triangleStart = triangleGeometry.list + i * cudaTriangleListSize();
        
        for(int j = 0; j < 3; ++j) {
            cudaTriangleListVX(triangleStart, j) = tri.p[i].x;
            cudaTriangleListVY(triangleStart, j) = tri.p[i].y;
            cudaTriangleListVZ(triangleStart, j) = tri.p[i].z;
        }
        
        cudaTriangleListPlaneA(triangleStart) = tri.plane.normal.x;
        cudaTriangleListPlaneB(triangleStart) = tri.plane.normal.y;
        cudaTriangleListPlaneC(triangleStart) = tri.plane.normal.z;
        cudaTriangleListPlaneD(triangleStart) = tri.plane.d;
        
        materials.list[i] = triangles.list[i].material;
        colors.list[i] = triangles.list[i].color;
    }
    
    deviceList.triangleGeometry = copyGeometryListToGPU(triangleGeometry);
    deviceList.triangleMaterials = copyGeometryListToGPU(materials);
    deviceList.triangleColors = copyGeometryListToGPU(colors);
    
    triangleGeometry.cleanup();
    materials.cleanup();
    colors.cleanup();
    
    return deviceList;
}

Scene createSceneOnDevice(Scene hostScene) {
    Scene deviceScene = hostScene;
    
    deviceScene.triangles = GeometryList<Triangle>(); //copyGeometryListToGPU(hostScene.triangles);
    deviceScene.spheres = copyGeometryListToGPU(hostScene.spheres);
    deviceScene.lights = copyGeometryListToGPU(hostScene.lights);
    
    deviceScene.cudaTriangles = convertTriangleGeometryListToCUDATriangleList(hostScene.triangles);
    
    return deviceScene;
}

Tracer* createRayTracerOnDevice(float angle, int screenW, int screenH, Scene scene, Color*& deviceFrameBuffer) {
    printf("Setting up renderer\n");
    Renderer deviceRenderer;
    deviceRenderer.initializeCuda(angle, screenW, screenH);
    
    printf("Setting up scene\n");
    Scene deviceScene = createSceneOnDevice(scene);
    
    Tracer hostTracer;
    hostTracer.renderer = deviceRenderer;
    hostTracer.scene = deviceScene;
    
    printf("Setting up raytracer\n");
    Tracer* deviceTracer;
    attemptCudaMalloc(&deviceTracer, sizeof(Tracer));
    
    hostTracer.tracer.scene = &deviceTracer->scene;
    
    hipMemcpy(deviceTracer, &hostTracer, sizeof(Tracer), hipMemcpyHostToDevice);
    
    deviceFrameBuffer = deviceRenderer.frameBuffer;
    
    printf("Done setting up device raytracer\n");
    
    return deviceTracer;
}

void copyFrameBufferFromDeviceToHost(Color* deviceFrameBuffer, Renderer& hostRenderer) {
    size_t size = sizeof(Color) * hostRenderer.screenW * hostRenderer.screenH;
    hipMemcpy(hostRenderer.frameBuffer, deviceFrameBuffer, size, hipMemcpyDeviceToHost);
}

void launchCudaKernel(float angle, int w, int h, Scene scene, Renderer& hostRenderer) {
    dim3 gridSize(w / BLOCK_WIDTH, h / BLOCK_HEIGHT, 1);
    dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT, 1);
    
    hipDeviceReset();
    
    Color* deviceFrameBuffer;
    Tracer* deviceTracer = createRayTracerOnDevice(angle, w, h, scene, deviceFrameBuffer);
    
    printf("Triangles on CPU: %d\n", scene.triangles.total);
    raytraceCudaKernel<<<gridSize, blockSize>>>(deviceTracer);
    hipDeviceSynchronize();
    
    copyFrameBufferFromDeviceToHost(deviceFrameBuffer, hostRenderer);
    
    cudaMemoryCleanup();
}


