#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <vector>

#include "Cuda.hpp"
#include "Vec3.hpp"
#include "Renderer.hpp"

using namespace std;

struct GPURayTracer {
    Scene* scene;
    
    CUDA_CALLABLE GPURayTracer(Scene* scene_) : scene(scene_) { }
    
    CUDA_DEVICE Intersection<CudaTriangleAttributes> findClosestIntersectedTriangle(const Ray& ray, const Shape* lastReflection) {
        CudaTriangleIntersection closestIntersection;
        CudaTriangleIntersection triIntersection;
        
        const int id = threadIdx.y * BLOCK_WIDTH + threadIdx.x;
        
        const int triangleSize = cudaTriangleListSize;
        
        __shared__ float trianglesGeometry[triangleSize * THREADS_IN_BLOCK];
        
        const int totalTrianglesInScene = scene->cudaTriangles.triangleGeometry.total / triangleSize;
        const int iterations = (totalTrianglesInScene  / THREADS_IN_BLOCK);
        
        const int trianglesInGroup = THREADS_IN_BLOCK;
        const int triangleGroupSize = triangleSize * trianglesInGroup;
        
        int closestId = -1;
        
        for(int i = 0; i < iterations; ++i) {
            for(int j = 0; j < triangleSize; ++j) {
                int trianglesGeometryPos    = id + j * trianglesInGroup;
                int sceneGeometryPos        = i * triangleGroupSize + trianglesGeometryPos;
                
                //if(id == 0 && blockIdx.x == 0 && blockIdx.y == 0)
                //    printf("Loaded %d\n", trianglesGeometryPos);
                
                trianglesGeometry[trianglesGeometryPos] = scene->cudaTriangles.triangleGeometry.list[sceneGeometryPos];
                
            }
            __syncthreads();
            
            for(int j = 0; j < trianglesInGroup; ++j) {
                CudaTriangle* triangle = (CudaTriangle*)(&trianglesGeometry[j * triangleSize]);
                
                if(triangle->calculateRayIntersections(ray, &triIntersection) > 0) {
                    int triId = i * trianglesInGroup + j;
                    if(triIntersection.distanceFromRayStartSquared < closestIntersection.distanceFromRayStartSquared) {
                        if(&scene->cudaTriangles.triangleAttributes.list[triId] == lastReflection) {
                            continue;
                        }
                        
                        
                        closestIntersection = triIntersection;
                        closestId = triId;
                        //printf("Intersection!\n");
                    }
                }
            }
            
            __syncthreads();
        }
   
        
   
   
//         for(Triangle* tri = scene->triangles.begin(); tri != scene->triangles.end(); ++tri) {
//             if(tri != lastReflection && tri->calculateRayIntersections(ray, &triIntersection) > 0) {
//                 //if(triIntersection.distanceFromRayStartSquared < closestIntersection.distanceFromRayStartSquared)
//                 //    closestIntersection = triIntersection;
//                 
//                 
//                 closestIntersection = minimum(triIntersection, closestIntersection);
//             }
//         }
        
        if(closestId == -1)
            return Intersection<CudaTriangleAttributes>();
        
        CudaTriangleAttributes* att = scene->cudaTriangles.triangleAttributes.list + closestId;
        Vec3 normal = att->calculateNormal(closestIntersection.intersectionS, closestIntersection.intersectionT);
        Intersection<CudaTriangleAttributes> inter(att, closestIntersection.pos, normal, closestIntersection.distanceFromRayStartSquared);
        
        return inter;
    }
    
    CUDA_DEVICE Intersection<Sphere> findClosestIntersectedSphere(const Ray& ray, const Shape* lastReflection) {
        Intersection<Sphere> closestIntersection;
        Intersection<Sphere> sphereIntersections[2];
        
        //printf("Total triangles: %d\n", (int)scene->triangles.total);
        
        for(Sphere* sphere = scene->spheres.begin(); sphere != scene->spheres.end(); ++sphere) {
            if(sphere == lastReflection)
                continue;
            
            int count = sphere->calculateRayIntersections(ray, sphereIntersections);
            
            if(count == 1) {
                closestIntersection = minimum(closestIntersection, sphereIntersections[0]);
            }
            else if(count == 2) {
                closestIntersection = minimum(closestIntersection, minimum(sphereIntersections[0], sphereIntersections[1]));
            }
        }
        
        return closestIntersection;
    }
    
    CUDA_DEVICE Intersection<Shape> findClosestIntersectedShape(const Ray& ray, const Shape* lastReflection) {
        return minimum(
            findClosestIntersectedTriangle(ray, lastReflection).toGenericShapeIntersection(),
            findClosestIntersectedSphere(ray, lastReflection).toGenericShapeIntersection()
        );
    }
};

typedef RayTracer<GPURayTracer> Tracer;

std::vector<void*> cudaAllocatedMemory;

void cudaMemoryCleanup() {
    for(int i = 0; i < cudaAllocatedMemory.size(); ++i) {
        printf("Freed %llx\n", (long long)cudaAllocatedMemory[i]);
        hipFree(cudaAllocatedMemory[i]);
    }
    
    hipDeviceReset();
}

template<typename T>
void attemptCudaMalloc(T dest, size_t size) {
    if(hipMalloc(dest, size) == hipErrorOutOfMemory) {
        cudaMemoryCleanup();
        throw "Failed to allocate memory on device";
    }
    
    cudaAllocatedMemory.push_back((void *)*dest);
}

void Renderer::initializeCuda(float angle, int w, int h) {
    screenW = w;
    screenH = h;
    
    attemptCudaMalloc(&frameBuffer, sizeof(Color) * w * h);
    
    viewAngle = angle;
    distToScreen = (w / 2 ) / tan(degToRadians(angle / 2));
}

__global__ void raytraceCudaKernel(Tracer* tracer) {
    //printf("Tracer dim: %d %d\n", tracer->renderer.screenW, tracer->renderer.screenH);
    
    //printf("Total triangles: %d\n", tracer->scene.triangles.total);
    
    //tracer->raytrace();
    
    int x = blockIdx.x * BLOCK_WIDTH + threadIdx.x;
    int y = blockIdx.y * BLOCK_HEIGHT + threadIdx.y;
    
    tracer->raytraceSingleRay(x, y);
}

template<typename T>
GeometryList<T> copyGeometryListToGPU(GeometryList<T> hostList) {
    GeometryList<T> deviceList;
    
    size_t size = sizeof(T) * hostList.total;
    
    printf("Transfered size: %d (%d)\n", (int)size, (int)sizeof(T));
    
    attemptCudaMalloc(&deviceList.list, size);
    hipMemcpy(deviceList.list, hostList.list, size, hipMemcpyHostToDevice);
    
    deviceList.total = hostList.total;
    
    return deviceList;
}

CUDATriangleList convertTriangleGeometryListToCUDATriangleList(GeometryList<Triangle>& triangles) {
    CUDATriangleList deviceList;
    
    GeometryList<float> triangleGeometry(triangles.total * cudaTriangleListSize);
    GeometryList<CudaTriangleAttributes> attributes(triangles.total);
    
    for(int i = 0; i < triangles.total; ++i) {
        Triangle& tri = triangles.list[i];
        float* triangleStart = triangleGeometry.list + i * cudaTriangleListSize;
        CudaTriangle* cudaTri = (CudaTriangle*)triangleStart;
        
        for(int j = 0; j < 3; ++j) {
            cudaTri->setVertex(j, tri.p[j]);
        }

        cudaTri->setPlane(tri.plane);
        
        attributes.list[i].color = triangles.list[i].color;
        attributes.list[i].material = triangles.list[i].material;
        attributes.list[i].normals[0] = triangles.list[i].normals[0];
        attributes.list[i].normals[1] = triangles.list[i].normals[1];
        attributes.list[i].normals[2] = triangles.list[i].normals[2];
    }
    
    deviceList.triangleGeometry = copyGeometryListToGPU(triangleGeometry);
    deviceList.triangleAttributes = copyGeometryListToGPU(attributes);
    
    triangleGeometry.cleanup();
    attributes.cleanup();
    
    return deviceList;
}

Scene createSceneOnDevice(Scene hostScene) {
    Scene deviceScene = hostScene;
    
    deviceScene.triangles = GeometryList<Triangle>(); //copyGeometryListToGPU(hostScene.triangles);
    deviceScene.spheres = copyGeometryListToGPU(hostScene.spheres);
    deviceScene.lights = copyGeometryListToGPU(hostScene.lights);
    
    deviceScene.cudaTriangles = convertTriangleGeometryListToCUDATriangleList(hostScene.triangles);
    
    return deviceScene;
}

Tracer* createRayTracerOnDevice(float angle, int screenW, int screenH, Scene scene, Color*& deviceFrameBuffer) {
    printf("Setting up renderer\n");
    Renderer deviceRenderer;
    deviceRenderer.initializeCuda(angle, screenW, screenH);
    
    printf("Setting up scene\n");
    Scene deviceScene = createSceneOnDevice(scene);
    
    Tracer hostTracer;
    hostTracer.renderer = deviceRenderer;
    hostTracer.scene = deviceScene;
    
    printf("Setting up raytracer\n");
    Tracer* deviceTracer;
    attemptCudaMalloc(&deviceTracer, sizeof(Tracer));
    
    hostTracer.tracer.scene = &deviceTracer->scene;
    
    hipMemcpy(deviceTracer, &hostTracer, sizeof(Tracer), hipMemcpyHostToDevice);
    
    deviceFrameBuffer = deviceRenderer.frameBuffer;
    
    printf("Done setting up device raytracer\n");
    
    return deviceTracer;
}

void copyFrameBufferFromDeviceToHost(Color* deviceFrameBuffer, Renderer& hostRenderer) {
    size_t size = sizeof(Color) * hostRenderer.screenW * hostRenderer.screenH;
    hipMemcpy(hostRenderer.frameBuffer, deviceFrameBuffer, size, hipMemcpyDeviceToHost);
}

void launchCudaKernel(float angle, int w, int h, Scene scene, Renderer& hostRenderer) {
    dim3 gridSize(w / BLOCK_WIDTH, h / BLOCK_HEIGHT, 1);
    dim3 blockSize(BLOCK_WIDTH, BLOCK_HEIGHT, 1);
    
    hipDeviceReset();
    
    Color* deviceFrameBuffer;
    Tracer* deviceTracer = createRayTracerOnDevice(angle, w, h, scene, deviceFrameBuffer);
    
    printf("Triangles on CPU: %d\n", scene.triangles.total);
    raytraceCudaKernel<<<gridSize, blockSize>>>(deviceTracer);
    hipDeviceSynchronize();
    
    copyFrameBufferFromDeviceToHost(deviceFrameBuffer, hostRenderer);
    
    cudaMemoryCleanup();
}


